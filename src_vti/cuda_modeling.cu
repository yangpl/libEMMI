/* CSEM FDTD modeling using Nvidia GPU
 *
 *   Copyright (c) 2020, Harbin Institute of Technology, China
 *   Author: Pengliang Yang
 *   E-mail: ypl.2100@gmail.com
 *   Homepage: https://yangpl.wordpress.com
 */
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>



#ifdef __cplusplus
extern "C" {
#endif
  
#include "cstd.h"
#include "acqui.h"
#include "emf.h"
#include "interp.h"
#include "constants.h"
#include "mpi_info.h"
  
void compute_green_function(emf_t *emf);
void extract_emf(acqui_t *acqui, emf_t *emf, interp_t *interp_rg, interp_t *interp_sg);
void write_data(acqui_t *acqui, emf_t *emf, char *fname, float _Complex ***dcal_fd);
  
#ifdef __cplusplus
}
#endif

#include "cuda_fdtd.cuh"


float c11, c21, c12, c22, c13, c23;

hipError_t status;
dim3 dimBlock, dimGrid, dimGrid_dft;

float *d_inveps11, *d_inveps22, *d_inveps33;
float *d_E1, *d_E2, *d_E3, *d_H1, *d_H2, *d_H3;
float *d_curlE1, *d_curlE2, *d_curlE3, *d_curlH1, *d_curlH2, *d_curlH3;
float *d_memD2E1, *d_memD3E1, *d_memD1E2, *d_memD3E2, *d_memD1E3, *d_memD2E3;
float *d_memD2H1, *d_memD3H1, *d_memD1H2, *d_memD3H2, *d_memD1H3, *d_memD2H3;
float *d_a1, *d_b1, *d_a2, *d_b2, *d_a3, *d_b3;
float *d_omegas;

hipFloatComplex *d_fwd_E1, *d_fwd_E2, *d_fwd_E3;
hipFloatComplex *d_backup, *d_expfactor;
int *d_corner_id, *h_ncorner;

hipfftHandle fftPlan;
hipFloatComplex *d_sH1kxky, *d_sH2kxky, *d_emfft, *d_emfft0;
float *d_sE12kxky;


int *d_rg_src_i1, *d_rg_src_i2, *d_rg_src_i3;
float *d_rg_src_w1, *d_rg_src_w2, *d_rg_src_w3;
int *d_sg_src_i1, *d_sg_src_i2, *d_sg_src_i3;
float *d_sg_src_w1, *d_sg_src_w2, *d_sg_src_w3;

int *d_rg_rec_i1, *d_rg_rec_i2, *d_rg_rec_i3;
float *d_rg_rec_w1, *d_rg_rec_w2, *d_rg_rec_w3;
int *d_sg_rec_i1, *d_sg_rec_i2, *d_sg_rec_i3;
float *d_sg_rec_w1, *d_sg_rec_w2, *d_sg_rec_w3;

int *d_chrec, *d_chsrc;
float *d_dres_td;//adjoint source


void cuda_fdtd_init(emf_t *emf)
{
  int ic, *h_chrec, *h_chsrc;
  int i1, i2, i3;

  int nchsrc = emf->nchsrc;
  int nchrec = emf->nchrec;
  int corner_id[8];

  //4-th order staggered FD, backward difference using shared memory:
  //c1*(D[0]-D[-1])+c2*(D[1]-D[-2])
  c11 = fd_c1/emf->d1;
  c21 = fd_c2/emf->d1;
  c12 = fd_c1/emf->d2;
  c22 = fd_c2/emf->d2;
  c13 = fd_c1/emf->d3;
  c23 = fd_c2/emf->d3;

  /* allocate memory on device */
  hipMalloc(&d_inveps11, emf->n123pad*sizeof(float));
  hipMalloc(&d_inveps22, emf->n123pad*sizeof(float));
  hipMalloc(&d_inveps33, emf->n123pad*sizeof(float));
  hipMalloc(&d_E1, emf->n123pad*sizeof(float));
  hipMalloc(&d_E2, emf->n123pad*sizeof(float));
  hipMalloc(&d_E3, emf->n123pad*sizeof(float));
  hipMalloc(&d_H1, emf->n123pad*sizeof(float));
  hipMalloc(&d_H2, emf->n123pad*sizeof(float));
  hipMalloc(&d_H3, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE1, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE2, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE3, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH1, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH2, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH3, emf->n123pad*sizeof(float));
  hipMalloc(&d_memD2E1, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3E1, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1E2, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3E2, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1E3, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2E3, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2H1, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3H1, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1H2, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3H2, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1H3, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2H3, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_fwd_E1, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_E2, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_E3, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_a1, emf->nb*sizeof(float));
  hipMalloc(&d_b1, emf->nb*sizeof(float));
  hipMalloc(&d_a2, emf->nb*sizeof(float));
  hipMalloc(&d_b2, emf->nb*sizeof(float));
  hipMalloc(&d_a3, emf->nb*sizeof(float));
  hipMalloc(&d_b3, emf->nb*sizeof(float));

  hipMalloc(&d_corner_id, 8*sizeof(int));
  hipHostAlloc(&h_ncorner, sizeof(int), hipHostMallocMapped);	
  hipMalloc(&d_backup, 8*sizeof(hipFloatComplex));
  hipMalloc(&d_expfactor, emf->nfreq*emf->nt*sizeof(hipFloatComplex));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - fdtd !\n"); exit(0); }
  
  //initialize memory on device
  hipMemcpy(d_inveps11, emf->inveps11[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_inveps22, emf->inveps22[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_inveps33, emf->inveps33[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemset(d_E1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_E2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_E3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_memD2E1, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3E1, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1E2, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3E2, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1E3, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2E3, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2H1, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3H1, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1H2, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3H2, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1H3, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2H3, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_fwd_E1, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_E2, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_E3, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemcpy(d_a1, emf->a1, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b1, emf->b1, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a2, emf->a2, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b2, emf->b2, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a3, emf->a3, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b3, emf->b3, emf->nb*sizeof(float), hipMemcpyHostToDevice);

  i1 = emf->nbe;
  i2 = emf->nbe;
  i3 = emf->nbe;
  corner_id[0] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe;
  i3 = emf->nbe;
  corner_id[1] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe;
  corner_id[2] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[3] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 -1;
  i2 = emf->nbe + emf->n2 -1;
  i3 = emf->nbe;
  corner_id[4] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[5] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[6] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[7] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  hipMemcpy(d_corner_id, corner_id, 8*sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_backup, 0, 8*sizeof(hipFloatComplex));
  hipMemcpy(d_expfactor, &emf->expfactor[0][0], emf->nfreq*emf->nt*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
  h_chsrc = alloc1int(nchsrc);
  h_chrec = alloc1int(nchrec);
  for(ic=0; ic<nchsrc; ++ic) {
    if     (strcmp(emf->chsrc[ic],"Ex")==0) h_chsrc[ic] = 1;
    else if(strcmp(emf->chsrc[ic],"Ey")==0) h_chsrc[ic] = 2;
    else if(strcmp(emf->chsrc[ic],"Ez")==0) h_chsrc[ic] = 3;
    else if(strcmp(emf->chsrc[ic],"Hx")==0) h_chsrc[ic] = 4;
    else if(strcmp(emf->chsrc[ic],"Hy")==0) h_chsrc[ic] = 5;
    else if(strcmp(emf->chsrc[ic],"Hz")==0) h_chsrc[ic] = 6;
  }
  for(ic=0; ic<nchrec; ++ic) {
    if     (strcmp(emf->chrec[ic],"Ex")==0) h_chrec[ic] = 1;
    else if(strcmp(emf->chrec[ic],"Ey")==0) h_chrec[ic] = 2;
    else if(strcmp(emf->chrec[ic],"Ez")==0) h_chrec[ic] = 3;
    else if(strcmp(emf->chrec[ic],"Hx")==0) h_chrec[ic] = 4;
    else if(strcmp(emf->chrec[ic],"Hy")==0) h_chrec[ic] = 5;
    else if(strcmp(emf->chrec[ic],"Hz")==0) h_chrec[ic] = 6;
  }
  hipMalloc(&d_chsrc, nchsrc*sizeof(int));
  hipMalloc(&d_chrec, nchrec*sizeof(int));
  hipMemcpy(d_chsrc, h_chsrc, nchsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_chrec, h_chrec, nchrec*sizeof(int), hipMemcpyHostToDevice);
  free(h_chsrc);
  free(h_chrec);

  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to initialize memory on device - fdtd !\n"); exit(0); }

}

void cuda_fdtd_close()
{
  /* free memory on device */
  hipFree(d_inveps11);
  hipFree(d_inveps22);
  hipFree(d_inveps33);
  hipFree(d_E1);
  hipFree(d_E2);
  hipFree(d_E3);
  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_H3);
  hipFree(d_curlE1);
  hipFree(d_curlE2);
  hipFree(d_curlE3);
  hipFree(d_curlH1);
  hipFree(d_curlH2);
  hipFree(d_curlH3);
  hipFree(d_memD2E1);
  hipFree(d_memD3E1);
  hipFree(d_memD1E2);
  hipFree(d_memD3E2);
  hipFree(d_memD1E3);
  hipFree(d_memD2E3);
  hipFree(d_memD2H1);
  hipFree(d_memD3H1);
  hipFree(d_memD1H2);
  hipFree(d_memD3H2);
  hipFree(d_memD1H3);
  hipFree(d_memD2H3);
  hipFree(d_fwd_E1);
  hipFree(d_fwd_E2);
  hipFree(d_fwd_E3);
  hipFree(d_a1);
  hipFree(d_b1);
  hipFree(d_a2);
  hipFree(d_b2);
  hipFree(d_a3);
  hipFree(d_b3);

  hipFree(d_corner_id);
  hipHostFree(h_ncorner);
  hipFree(d_backup);
  hipFree(d_expfactor);
  
  hipFree(d_chsrc);
  hipFree(d_chrec);

}


void cuda_airwave_bc_init(emf_t *emf)
{
  int n1fft = emf->n1fft;
  int n2fft = emf->n2fft;
  
  // create FFT plan
  hipfftPlan2d(&fftPlan, n1fft, n2fft, HIPFFT_C2C);
  hipMalloc(&d_sH1kxky, n1fft*n2fft*emf->rd*sizeof(hipFloatComplex));
  hipMalloc(&d_sH2kxky, n1fft*n2fft*emf->rd*sizeof(hipFloatComplex));
  hipMalloc(&d_sE12kxky, n1fft*n2fft*(emf->rd-1)*sizeof(float));
  hipMalloc(&d_emfft, n1fft*n2fft*sizeof(hipFloatComplex));
  hipMalloc(&d_emfft0, n1fft*n2fft*sizeof(hipFloatComplex));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - airwave !\n"); exit(0); }

  
  hipMemcpy(d_sH1kxky, &emf->sH1kxky[0][0][0], n1fft*n2fft*emf->rd*sizeof(float _Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_sH2kxky, &emf->sH2kxky[0][0][0], n1fft*n2fft*emf->rd*sizeof(float _Complex), hipMemcpyHostToDevice);
  if(emf->rd>1) hipMemcpy(d_sE12kxky, &emf->sE12kxky[0][0][0], n1fft*n2fft*(emf->rd-1)*sizeof(float), hipMemcpyHostToDevice);
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to initialize memory on device - airwave!\n"); exit(0); }
  
}


void cuda_airwave_bc_close()
{
  hipfftDestroy(fftPlan);
  hipFree(d_sH1kxky);
  hipFree(d_sH2kxky);
  hipFree(d_sE12kxky);
  hipFree(d_emfft);
  hipFree(d_emfft0);
  
}



void cuda_interpolation_init(acqui_t *acqui, emf_t *emf, interp_t *interp_rg, interp_t *interp_sg)
{
  int rd = emf->rd;

  hipMalloc(&d_rg_src_i1, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_i2, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_i3, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_w1, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_rg_src_w2, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_rg_src_w3, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));

  hipMalloc(&d_sg_src_i1, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_i2, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_i3, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_w1, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_sg_src_w2, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_sg_src_w3, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  
  hipMalloc(&d_rg_rec_i1, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_rg_rec_i2, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_rg_rec_i3, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_rg_rec_w1, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));
  hipMalloc(&d_rg_rec_w2, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));
  hipMalloc(&d_rg_rec_w3, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));

  hipMalloc(&d_sg_rec_i1, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_sg_rec_i2, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_sg_rec_i3, acqui->nrec*acqui->nsubrec*sizeof(int));
  hipMalloc(&d_sg_rec_w1, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));
  hipMalloc(&d_sg_rec_w2, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));
  hipMalloc(&d_sg_rec_w3, 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - interpolation!\n"); exit(0);  }

  //-------------------------------------------------------------------------
  hipMemcpy(d_rg_src_i1, interp_rg->src_i1[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_i2, interp_rg->src_i2[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_i3, interp_rg->src_i3[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w1, interp_rg->src_w1[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w2, interp_rg->src_w2[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w3, interp_rg->src_w3[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_sg_src_i1, interp_sg->src_i1[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_i2, interp_sg->src_i2[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_i3, interp_sg->src_i3[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w1, interp_sg->src_w1[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w2, interp_sg->src_w2[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w3, interp_sg->src_w3[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_rg_rec_i1, interp_rg->rec_i1[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_rec_i2, interp_rg->rec_i2[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_rec_i3, interp_rg->rec_i3[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_rec_w1, interp_rg->rec_w1[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_rec_w2, interp_rg->rec_w2[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_rec_w3, interp_rg->rec_w3[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_sg_rec_i1, interp_sg->rec_i1[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_rec_i2, interp_sg->rec_i2[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_rec_i3, interp_sg->rec_i3[0], acqui->nrec*acqui->nsubrec*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_rec_w1, interp_sg->rec_w1[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_rec_w2, interp_sg->rec_w2[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_rec_w3, interp_sg->rec_w3[0][0], 2*rd*acqui->nrec*acqui->nsubrec*sizeof(float), hipMemcpyHostToDevice);

  status = hipGetLastError();
  if(hipSuccess!=status) { printf("Failed to initialize memory on device - interpolation !\n"); exit(0); }
}

void cuda_interpolation_close()
{
  hipFree(d_rg_src_i1);
  hipFree(d_rg_src_i2);
  hipFree(d_rg_src_i3);
  hipFree(d_rg_src_w1);
  hipFree(d_rg_src_w2);
  hipFree(d_rg_src_w3);

  hipFree(d_sg_src_i1);
  hipFree(d_sg_src_i2);
  hipFree(d_sg_src_i3);
  hipFree(d_sg_src_w1);
  hipFree(d_sg_src_w2);
  hipFree(d_sg_src_w3);

  hipFree(d_rg_rec_i1);
  hipFree(d_rg_rec_i2);
  hipFree(d_rg_rec_i3);
  hipFree(d_rg_rec_w1);
  hipFree(d_rg_rec_w2);
  hipFree(d_rg_rec_w3);

  hipFree(d_sg_rec_i1);
  hipFree(d_sg_rec_i2);
  hipFree(d_sg_rec_i3);
  hipFree(d_sg_rec_w1);
  hipFree(d_sg_rec_w2);
  hipFree(d_sg_rec_w3);
}


extern "C"
void cuda_modeling(acqui_t *acqui, emf_t *emf, interp_t *interp_rg, interp_t *interp_sg, int adj)
{
  double t_update_E,t_update_H,t_inject_E,t_inject_H,t_curlE, t_curlH,t_dft_emf,t0,t_convergence;
  static int doneinit = 0;

  if(!doneinit){
    hipSetDevice(0);// initialize device, default device=0;
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to initialize device!\n"); exit(0); }
    doneinit = 1;
  }

  /*==========================================================*/
  cuda_interpolation_init(acqui, emf, interp_rg, interp_sg);
  cuda_fdtd_init(emf);
  if(emf->airwave) cuda_airwave_bc_init(emf);

  if(adj){
    hipMalloc(&d_dres_td, emf->nt*acqui->nrec*emf->nchrec*sizeof(float));
    hipMemcpy(d_dres_td, &emf->dres_td[0][0][0], emf->nt*acqui->nrec*emf->nchrec*sizeof(float), hipMemcpyHostToDevice);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to allocate memory on device - fdtd !\n"); exit(0); }
    
  }
    
  dimBlock.x = BlockSize1; 
  dimBlock.y = BlockSize2;
  dimGrid.x = (emf->n1pad+BlockSize1-1)/BlockSize1;
  dimGrid.y = (emf->n2pad+BlockSize2-1)/BlockSize2;
  dimGrid_dft.x = (emf->n1fft+BlockSize1-1)/BlockSize1;
  dimGrid_dft.y = (emf->n2fft+BlockSize2-1)/BlockSize2;
  if(emf->verb){
    printf("dimBlock.x=%d \n", dimBlock.x);
    printf("dimBlock.y=%d \n", dimBlock.y);
    printf("dimGrid.x=%d \n", dimGrid.x);
    printf("dimGrid.y=%d \n", dimGrid.y);
    printf("dimGrid_dft.x=%d \n", dimGrid_dft.x);
    printf("dimGrid_dft.y=%d \n", dimGrid_dft.y);

    t0 = 0;
    t_curlE = 0.;
    t_inject_H = 0.;
    t_update_H = 0.;
    t_curlH = 0.;
    t_inject_E = 0.;
    t_update_E = 0.;
    t_dft_emf= 0.;
    t_convergence=0.;
  }

  int it;
  float mstimer;
  hipEvent_t start, stop;

  hipEventCreate(&start);	
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(it=0; it<emf->nt; it++){
    if(emf->verb && it%50==0) printf("it---- %d\n", it);

    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_curlE<<<dimGrid,dimBlock>>>
      (d_E1, d_E2, d_E3, d_curlE1, d_curlE2, d_curlE3, d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
       d_memD1E2, d_memD1E3, d_memD2E1, d_memD2E3, d_memD3E1, d_memD3E2, 
       c11, c21, c12, c22, c13, c23,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave, 
       adj?emf->i3min_adj[it]:emf->i3min_fwd[it], adj?emf->i3max_adj[it]:emf->i3max_fwd[it]);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute curlE on device!\n"); exit(0); }    
    if(emf->verb) t_curlE += MPI_Wtime()-t0;

    
    if(emf->verb) t0 = MPI_Wtime();
    cuda_inject_magnetic_source<<<(emf->nchsrc*acqui->nsrc+BlockSize-1)/BlockSize,BlockSize>>>
      (d_rg_src_i1, d_rg_src_i2, d_rg_src_i3, d_rg_src_w1, d_rg_src_w2, d_rg_src_w3,
       d_sg_src_i1, d_sg_src_i2, d_sg_src_i3, d_sg_src_w1, d_sg_src_w2, d_sg_src_w3,
       d_curlE1, d_curlE2, d_curlE3, d_chsrc,
       emf->stf[it], emf->d1, emf->d2, emf->d3, emf->nchsrc, acqui->nsrc, acqui->nsubsrc,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->nbe, emf->rd);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to inject magnetic source on device!\n"); exit(0); }
    if(emf->verb) t_inject_H += MPI_Wtime()-t0;

    
    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_update_H<<<dimGrid,dimBlock>>>(d_H1, d_H2, d_H3, d_curlE1, d_curlE2, d_curlE3,
					     emf->dt, emf->n1pad, emf->n2pad, emf->n3pad);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to update H on device!\n"); exit(0); }    
    if(emf->airwave){
      cuda_airwave_bc_copy<<<dimGrid_dft,dimBlock>>>
    	(d_emfft, &d_H3[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft0, HIPFFT_FORWARD);//FFT into wavenumber domain

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dft,dimBlock>>>(d_emfft, d_sH1kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dft,dimBlock>>>
	(&d_H1[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dft,dimBlock>>>(d_emfft, &d_sH1kxky[emf->n1fft*emf->n2fft], emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dft,dimBlock>>>
	(&d_H1[emf->n1pad*emf->n2pad*(emf->nbe-2)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dft,dimBlock>>>(d_emfft, d_sH2kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dft,dimBlock>>>
	(&d_H2[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dft,dimBlock>>>(d_emfft, &d_sH2kxky[emf->n1fft*emf->n2fft], emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dft,dimBlock>>>
	(&d_H2[emf->n1pad*emf->n2pad*(emf->nbe-2)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      status = hipGetLastError();
      if (hipSuccess!=status) { printf("Failed to handle air-water interface!\n"); exit(0); }
    }
    if(emf->verb) t_update_H += MPI_Wtime()-t0;


    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_curlH<<<dimGrid,dimBlock>>>(d_H1, d_H2, d_H3, d_curlH1, d_curlH2, d_curlH3,
    					  d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
    					  d_memD1H2, d_memD1H3, d_memD2H1,
    					  d_memD2H3, d_memD3H1, d_memD3H2, 
    					  d_inveps11, d_inveps22, d_inveps33,
    					  c11, c21, c12, c22, c13, c23,
    					  emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave,
					  adj?emf->i3min_adj[it]:emf->i3min_fwd[it], adj?emf->i3max_adj[it]:emf->i3max_fwd[it]);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute curlH on device!\n"); exit(0); }    
    if(emf->verb) t_curlH += MPI_Wtime()-t0;

    if(emf->verb) t0 = MPI_Wtime();
    if(adj){
      cuda_inject_electric_src_adj<<<(emf->nchrec*acqui->nrec+BlockSize-1)/BlockSize,BlockSize>>>
	(d_rg_rec_i1, d_rg_rec_i2, d_rg_rec_i3, d_rg_rec_w1, d_rg_rec_w2, d_rg_rec_w3,
	 d_sg_rec_i1, d_sg_rec_i2, d_sg_rec_i3, d_sg_rec_w1, d_sg_rec_w2, d_sg_rec_w3,
	 d_inveps11, d_inveps22, d_inveps33, d_curlH1, d_curlH2, d_curlH3, d_chrec, 
	 d_dres_td, it, emf->nt, emf->nchrec, acqui->nrec, acqui->nsubrec,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nbe, emf->rd);
    }else{
      cuda_inject_electric_src_fwd<<<(emf->nchsrc*acqui->nsrc+BlockSize-1)/BlockSize,BlockSize>>>
	(d_rg_src_i1, d_rg_src_i2, d_rg_src_i3, d_rg_src_w1, d_rg_src_w2, d_rg_src_w3,
	 d_sg_src_i1, d_sg_src_i2, d_sg_src_i3, d_sg_src_w1, d_sg_src_w2, d_sg_src_w3,
	 d_inveps11, d_inveps22, d_inveps33, d_curlH1, d_curlH2, d_curlH3, d_chsrc, 
	 emf->stf[it], emf->d1, emf->d2, emf->d3, emf->nchsrc, acqui->nsrc, acqui->nsubsrc,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nbe, emf->rd);
    }
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to inject electric source on device!\n"); exit(0); }
    if(emf->verb) t_inject_E += MPI_Wtime()-t0;

    
    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_update_E<<<dimGrid,dimBlock>>>
      (d_E1, d_E2, d_E3, d_curlH1, d_curlH2, d_curlH3, d_inveps11, d_inveps22, d_inveps33,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->dt);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to update E on device!\n"); exit(0); }    
    if(emf->airwave){
      cuda_airwave_bc_copy<<<dimGrid_dft,dimBlock>>>
	(d_emfft, &d_E1[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_FORWARD);//FFT into wavenumber domain
      cuda_airwave_bc_scale_FE<<<dimGrid_dft,dimBlock>>>(d_emfft, d_sE12kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid,dimBlock>>>
    	(&d_E1[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      cuda_airwave_bc_copy<<<dimGrid_dft,dimBlock>>>
	(d_emfft, &d_E2[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_FORWARD);//FFT into wavenumber domain
      cuda_airwave_bc_scale_FE<<<dimGrid_dft,dimBlock>>>(d_emfft, d_sE12kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid,dimBlock>>>
    	(&d_E2[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);

      status = hipGetLastError();
      if (hipSuccess!=status) { printf("Failed to handle air-water interface!\n"); exit(0); }
    }
    if(emf->verb) t_update_E += MPI_Wtime()-t0;
    
    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_E1, d_fwd_E2, d_fwd_E3, &d_expfactor[it*emf->nfreq], d_E1, d_E2, d_E3, 
					emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq,
					adj?emf->i3min_adj[it]:emf->i3min_fwd[it], adj?emf->i3max_adj[it]:emf->i3max_fwd[it]);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute DFT of E + H on device!\n"); exit(0); }    
    if(emf->verb) t_dft_emf += MPI_Wtime()-t0;


    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    if(it%100==0){/* convergence check */
      cuda_check_convergence1<<<1,8>>>(d_corner_id, d_fwd_E1, d_backup, h_ncorner);
      if(emf->verb) printf("%d corners of the cube converged!\n", h_ncorner[0]);
      if(h_ncorner[0]==8) { emf->nt = it; printf("converge after %d steps\n", it); break; }/* all 8 corners converged, exit now */
    }
    if(emf->verb) t_convergence += MPI_Wtime()-t0;
  }

  
  if(emf->verb) {
    t0 = t_curlH + t_inject_E + t_update_E + t_curlE + t_inject_H + t_update_H
      + t_dft_emf + t_convergence;
    printf("-------------- elapsed time --------------------\n");
    printf("    compute curlE: %e s\n", t_curlE);
    printf("    inject magnetic source: %e s\n", t_inject_H);
    printf("    update magnetic field: %e s\n", t_update_H);

    printf("    compute curlH: %e s\n", t_curlH);
    printf("    inject electric source: %e s\n", t_inject_E);
    printf("    update electric field: %e s\n", t_update_E);
    
    printf("    DFT EM field: %e s\n", t_dft_emf);
    printf("    convergence check: %e s\n", t_convergence);
    printf("    Total modeling time: %e s\n", t0);
    printf("------------------------------------------------\n");
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&mstimer, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if(emf->verb) printf("Elapsed time: %g (s)\n", mstimer*1.e-3);

  if(adj){
    hipMemcpy(&emf->adj_E1[0][0][0][0], &d_fwd_E1[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&emf->adj_E2[0][0][0][0], &d_fwd_E2[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&emf->adj_E3[0][0][0][0], &d_fwd_E3[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }else{
    hipMemcpy(&emf->fwd_E1[0][0][0][0], &d_fwd_E1[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&emf->fwd_E2[0][0][0][0], &d_fwd_E2[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&emf->fwd_E3[0][0][0][0], &d_fwd_E3[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }

  cuda_interpolation_close();
  cuda_fdtd_close(); 
  if(emf->airwave) cuda_airwave_bc_close();

  if(adj) hipFree(d_dres_td);
}
